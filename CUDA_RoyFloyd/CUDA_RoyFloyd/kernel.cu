#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#define DIMENSION 5;
#define INF 999999

typedef struct {
	int rows;
	int cols;
	float* elements;
}Matrix;

//typedef struct Matrix Matrix;

//void Matrix* GenMatrix(int rows, int cols)
//{
//	struct Matrix matrix;
//	matrix.rows = rows;
//	matrix.cols = cols;
//	matrix.elements = (double**)malloc(sizeof(double*)* rows);
//	for (int x = 0; x < rows; x++){
//		matrix.elements[x] = (double*)calloc(cols, sizeof(double));
//	}
//	struct Matrix *m;
//	m = &matrix;
//	return m;
//}

__global__ void RoyFloyd(Matrix M,int k)
{
	int i,j, k;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (k = 0; k < DIMENSION; k++)
	{
		for (i = 0; i < M; i++)
		{
			for (j = 0; j < M; j++){
				if (M.elements[row*M.rows + k] + M.elements[k*A.rows + col] < M.elements[row*M.rows + k])
				{
					M.elements[row*M.rows + col] = M.elements[row*M.rows + k] + M.elements[k*M.rows + col];
				}
			}	
		}
	}
	__syncthreads();
}


int main(int argc, char **argv)
{
	Matrix M;
	M.rows = DIMENSION;
    M.cols = DIMENSION;

	size_t size = M.rows * M.cols * sizeof(float);
	M.elements = (float*)malloc(size);

	M.elements[0] = 0;
	M.elements[1] = 2;
	M.elements[2] = INF;
	M.elements[3] = 10;
	M.elements[4] = INF;
	M.elements[5] = 2;
	M.elements[6] = 0;
	M.elements[7] = 3;
	M.elements[8] = INF;
	M.elements[9] = INF;
	M.elements[10] = INF;
	M.elements[11] = 3;
	M.elements[12] = 0;
	M.elements[13] = 1;
	M.elements[14] = 8;
	M.elements[15] = 10;
	M.elements[16] = INF;
	M.elements[17] = 1;
	M.elements[18] = 0;
	M.elements[19] = INF;
	M.elements[20] = INF;
	M.elements[21] = INF;
	M.elements[22] = 8;
	M.elements[23] = INF;
	M.elements[24] = 0;
	

	// Allocate C in device memory
	Matrix N;
	N.rows = N.rows;
	N.cols = N.cols;
	size_t size = N.rows * N.cols * sizeof(float);
	hipMalloc(&N.elements, size);

	// Read C from device memory
	hipMemcpy(N.elements, M.elements, size, hipMemcpyHostToDevice);

	// Invoke kernel
	dim3 dimBlock(DIMENSION, DIMENSION);
	dim3 dimGrid(N.rows / dimBlock.x, N.cols / dimBlock.y);

	RoyFloyd << <dimGrid, dimBlock >> >(N, k);
	hipMemcpy(M.elements, M.elements, size, hipMemcpyDeviceToHost);	
	hipMemcpy(N.elements, M.elements, size, hipMemcpyHostToDevice);

	hipFree(N.elements);
	free(M.elements);
}